#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
using namespace std;
#include <stdio.h>
#include "cutil_math.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <unordered_map>

#define M_PI 3.14159265358979323846;

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA error: %s: %s. \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// hash function to calculate new seed for each frame
// see http://www.reedbeta.com/blog/2013/01/12/quick-and-easy-gpu-random-numbers-in-d3d11/
uint WangHash(uint a) {
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

// "__host__": This function called by CPU and runs on CPU
// "__device__": This function called by GPU and runs on GPU (inside one thread)
// "__global__": This is a kernel function, called by CPU and runs on GPU
// "__constant__": This data won't and can't be modified

// Changing variables
__constant__ float3 camPos = { 30.0f, 0.0f, 300.0f };	// -left, +right
#define USING_WAVE 0	// from 0 to 10

// reflection type (DIFFuse, SPECular, REFRactive)
enum Refl_t { DIFF, SPEC, REFR };
// geometry type
enum Geom_t { SPHERE, CONE };

// mat name
#define mat_human 0
#define mat_marble 1
#define mat_paint 2
#define mat_glass 3
#define mat_rubber 4
#define mat_brass 5
#define mat_road 6
#define mat_al 7
#define mat_al2o3 8
#define mat_brick 9

__constant__ float wave[11] = {
	7.8576538e+02,
	8.1770000e+02,
	8.6250000e+02,
	9.1025000e+02,
	9.4255000e+02,
	9.7750000e+02,
	1.0277500e+03,
	1.0780000e+03,
	1.1255000e+03,
	1.1860000e+03,
	1.2766667e+03
};


// emiLib[matName][waveNum]
__constant__ float emiLib[10][11] = {
	9.9000000e-01,	9.5834758e-01,	8.7470001e-01,	5.0455443e-01,	9.2789246e-01,	1.2250251e-01,	9.6426578e-01,	5.5701898e-01,	4.1617280e-02,	9.7773773e-01,
	9.9000000e-01,	9.5462609e-01,	8.8365367e-01,	2.8523451e-01,	9.2827028e-01,	1.1789014e-01,	9.7194589e-01,	5.4616836e-01,	4.1602933e-02,	9.7348785e-01,
	9.9000000e-01,	9.5099592e-01,	9.6279529e-01,	3.8887318e-01,	9.2640468e-01,	1.2078545e-01,	9.6430868e-01,	5.2990503e-01,	4.0821044e-02,	9.6252597e-01,
	9.9000000e-01,	9.5741246e-01,	8.6909910e-01,	4.2252257e-01,	9.2027605e-01,	1.2892990e-01,	9.4494491e-01,	5.1621436e-01,	4.8036999e-02,	9.4693874e-01,
	9.9000000e-01,	9.6385735e-01,	8.5889954e-01,	4.4505789e-01,	9.2317386e-01,	1.3452107e-01,	9.5513005e-01,	5.0484414e-01,	1.4619579e-01,	9.3275042e-01,
	9.9000000e-01,	9.6087765e-01,	9.3344199e-01,	4.7704424e-01,	8.9968776e-01,	1.4311263e-01,	9.5631467e-01,	4.9568769e-01,	2.6974721e-01,	9.1201603e-01,
	9.9000000e-01,	9.5962251e-01,	9.4205163e-01,	5.6399482e-01,	8.6774658e-01,	1.4932587e-01,	9.5258259e-01,	4.7984848e-01,	4.2480553e-01,	8.7901868e-01,
	9.9000000e-01,	9.5305901e-01,	9.4627694e-01,	3.2859562e-01,	8.8061124e-01,	1.4229701e-01,	9.1783893e-01,	4.6578646e-01,	4.7823023e-01,	8.5128884e-01,
	9.9000000e-01,	9.5385122e-01,	9.5199753e-01,	4.2369253e-02,	8.9911606e-01,	1.3455656e-01,	9.1771733e-01,	4.5454008e-01,	5.1389488e-01,	9.0261137e-01,
	9.9000000e-01,	9.5852822e-01,	9.5649050e-01,	2.7487807e-02,	9.1817783e-01,	1.2604779e-01,	9.1884949e-01,	4.3838823e-01,	5.4462383e-01,	9.3754130e-01,
	9.9000000e-01,	9.5240096e-01,	9.5069231e-01,	8.9005827e-02,	9.3104627e-01,	1.1098321e-01,	9.5362853e-01,	4.1783501e-01,	5.6727138e-01,	9.7270040e-01
};

__device__ float BBp (float T, float v)
{
	// 2e8*2*pi*h_bar*c^2
	double c1 = 1.1910429524674593e-08;
	// 100*2*pi*h_bar*c/k
	double c2 = 1.4387773536379256;
	// BBp = c1*pow(v,3)/(exp(c2*v/T)-1)
	return float(c1 * pow(v, 3) / (exp(c2 * v / T) - 1));
}

struct Ray {
	float3 origin;
	float3 direction;
	// create a ray
	__device__ Ray(float3 o_, float3 d_) : origin(o_), direction(d_) {}
};

struct Hit
{
	float hitDist;		//hitDistance
	float3 normal;
	float3 oriNormal;	// oriented normal (for rafraction)
	float3 nextDir;		// direction for next segment
	int matName;
	float temperature;
	float emi;	// 1 - emiLib[matName("matName")][waveNum("wave_1")]
	Refl_t reflectType;
	Geom_t geomtryType;
	int geomID;
	__device__ void Init() {
		hitDist = 1e20;
		normal = make_float3(0.0f);
		oriNormal = make_float3(0.0f);
		nextDir = make_float3(0.0f);
		matName = -1;
		temperature = 0.0f;
		emi = 0.0f;
		reflectType = DIFF;
		geomtryType = SPHERE;
		geomID = -1;
	}
};

struct Sphere {

	float radius;
	float3 position;
	int matName;
	float temperature;
	Refl_t reflectType;	//DIFF, SPEC, REFR
	__device__ float intersect(const Ray& ray) const { // returns distance, 0 if nohit 

		// Ray/sphere intersection
		// Quadratic formula required to solve ax^2 + bx + c = 0 
		// Solution x = (-b +- sqrt(b*b - 4ac)) / 2a
		// Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 

		float3 op = position - ray.origin;
		float t, epsilon = 0.01f;
		float b = dot(op, ray.direction);
		float disc = b * b - dot(op, op) + radius * radius; // discriminant
		if (disc < 0) return 0; else disc = sqrtf(disc);
		return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
	}
};

struct Cone {
	float3 tip, axis;
	float cosA, height;
	int matName;
	float temperature;
	Refl_t reflectType;	//DIFF, SPEC, REFR
	__device__ float intersect(const Ray& ray) const { // returns distance, 0 if nohit  

		float3 co = ray.origin - tip; float cos2t = cosA; cos2t *= cos2t;
		float t, dotDV = dot(ray.direction, axis), dotCOV = dot(co, axis);
		float a = dotDV * dotDV - cos2t, b = 2.0f * (dotDV * dotCOV - dot(ray.direction, co) * cos2t),
			c = dotCOV * dotCOV - dot(co, co) * cos2t, delta = b * b - 4 * a * c;
		if (delta <= 0.0f) return 0; else delta = sqrt(delta);
		t = (-b + delta) / 2.0f / a > 0.01f ? (-b + delta) / 2.0f / a : max((-b - delta) / 2.0f / a, 0.0f);
		float3 hit = ray.origin + t * ray.direction;
		if (dot(hit - tip, axis) <= 0.0f) return 0;
		return t;
	}
};

__constant__ Sphere spheres[] = {
	/* cornell box
	{radius	position						matName		temperature			reflectType*/
	{1e5f,	{-1e5f - 100.0f, 0.0f, 0.0f},	mat_brick,	20.0f + 273.15f,	DIFF},// left wall
	{1e5f,	{1e5f + 100.0f, 0.0f, 0.0f},	mat_brick,	20.0f + 273.15f,	DIFF},// right wall
	{1e5f,	{0.0f, 0.0f, -1e5f - 100.0f},	mat_brick,	20.0f + 273.15f,	DIFF},// back wall
	{1e5f,	{0.0f, 0.0f, 1e5f + 500.0f},	mat_brick,	20.0f + 273.15f,	DIFF},// front wall
	{1e5f,	{0.0f, -1e5f - 100.0f, 0.0f},	mat_road,	20.0f + 273.15f,	DIFF},// floor
	{1e5f,	{0.0f, 1e5f + 100.0f, 0.0f},	mat_brick,	20.0f + 273.15f,	DIFF},// ceiling  
	{40.0f,	{50.0f ,-70.0f, 0.0f},			mat_al,		72.5f + 273.15f,	DIFF},// sphere 
	{50.0f,	{0.0f ,135.0f, 0.0f},			mat_glass,	100.0f + 273.15f,	DIFF} // lamp 
};

__constant__ Cone cones[] = {
	/*
	tip							axis					cosA	height	matName		temperature			reflectType*/
	{{-50.0f, -20.0f, -80.0f},	{0.0f, -1.0f, 0.0f},	0.976296f,	80.0f,	mat_rubber,	37.0f + 273.15f,	DIFF}
};

__device__ inline bool intersect_scene(const Ray& ray, Hit& bestHit)
{
	float d = 1e20;
	float INF = 1e20;

	// intersect all spheres in the scene
	float spheresNum = sizeof(spheres) / sizeof(Sphere);
	for (int i = 0; i < spheresNum; i++)  // for all spheres in scene
	{
		// keep track of distance from origin to closest intersection point
		if ((d = spheres[i].intersect(ray)) && d < bestHit.hitDist && d > 0)
		{ 
			bestHit.hitDist = d;
			bestHit.geomtryType = SPHERE;
			bestHit.geomID = i;
		}
	}

	// intersect all cones in the scene
	float conesNum = sizeof(cones) / sizeof(Cone);
	for (int i = 0; i < conesNum; i++)  // for all cones in scene
	{
		// keep track of distance from origin to closest intersection point
		if ((d = cones[i].intersect(ray)) && d < bestHit.hitDist && d > 0)
		{
			bestHit.hitDist = d;
			bestHit.geomtryType = CONE;
			bestHit.geomID = i;
		}
	}

	// t is distance to closest intersection of ray with all primitives in the scene
	if (bestHit.hitDist < INF)
	{
		float3 hitPostion = ray.origin + ray.direction * bestHit.hitDist;
		switch (bestHit.geomtryType)
		{
		case SPHERE:
			bestHit.normal = normalize(hitPostion - spheres[bestHit.geomID].position);
			bestHit.oriNormal = dot(bestHit.normal, ray.direction) < 0.0f ? bestHit.normal : bestHit.normal * -1.0f;
			bestHit.matName = spheres[bestHit.geomID].matName;
			bestHit.temperature = spheres[bestHit.geomID].temperature;
			bestHit.emi = emiLib[bestHit.matName][USING_WAVE];	// start from 0 
			bestHit.reflectType = spheres[bestHit.geomID].reflectType;
			break;
		case CONE:
			float3 cp = hitPostion - cones[bestHit.geomID].tip;
			bestHit.normal = normalize(cp * dot(cones[bestHit.geomID].axis, cp) / dot(cp, cp) - cones[bestHit.geomID].axis);
			bestHit.oriNormal = dot(bestHit.normal, ray.direction) < 0.0f ? bestHit.normal : bestHit.normal * -1.0f;
			bestHit.matName = cones[bestHit.geomID].matName;
			bestHit.temperature = cones[bestHit.geomID].temperature;
			bestHit.emi = emiLib[bestHit.matName][USING_WAVE];	// start from 0 
			bestHit.reflectType = cones[bestHit.geomID].reflectType;
			break;
		default:
			break;
		}
		return true;
	}
	else return false;
}

// radiance function
// compute path bounces in scene and accumulate returned color from each path sgment
__device__ float radiance(Ray& ray, hiprandState* randstate, int frameNum) { // returns ray color

	Hit bestHit;
	// color mask
	float mask = 1.0f;
	// accumulated color for current pixel
	float accuIntensity = 0.0f;

	//// hit debug
	//bestHit.Init();
	//if (!intersect_scene(ray, bestHit))
	//	return 0.0f; // if miss, return black
	//else
	//{
	//	return bestHit.temperature/500.f;
	//	//return bestHit.emission;
	//}
	//// hit debug end
	

	int bounces = 0;
	while(bounces < 5 || hiprand_uniform(randstate) < 0.5f)
	{  
		bounces++;
		bestHit.Init();
		// intersect ray with scene
		if (!intersect_scene(ray, bestHit))
			return 0.0f; // if miss, return black
		// else: we've got a hit with a scene primitive
		accuIntensity += (mask * BBp(bestHit.temperature, wave[USING_WAVE])*bestHit.emi);
		float3 hitPosition = ray.origin + ray.direction * bestHit.hitDist;

		// SHADING: diffuse, specular or refractive

		// ideal diffuse reflection
		if (bestHit.reflectType == DIFF)
		{
			// create 2 random numbers
			float r1 = 2 * 3.1415926 * hiprand_uniform(randstate);
			float r2 = hiprand_uniform(randstate);
			float r2s = sqrtf(r2);

			// compute orthonormal coordinate frame uvw with hitpoint as origin 
			float3 w = bestHit.oriNormal;
			float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = cross(w, u);

			// compute cosine weighted random ray direction on hemisphere 
			bestHit.nextDir = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1 - r2));

			// offset origin next path segment to prevent self intersection
			hitPosition += bestHit.oriNormal * 0.03;

			// multiply mask with color of object
			mask *= 1.0f - bestHit.emi;
		}

		// ideal specular reflection
		if (bestHit.reflectType == SPEC)
		{

			// reflect
			bestHit.nextDir = ray.direction - 2.0f * bestHit.normal * dot(bestHit.normal, ray.direction);

			// offset origin next path segment to prevent self intersection
			hitPosition += bestHit.oriNormal * 0.01;

			// multiply color to the object
			mask *= 1.0f - bestHit.emi;
		}

		// ideal refraction (based on smallpt code by Kevin Beason)
		if (bestHit.reflectType == REFR)
		{

			bool into = dot(bestHit.normal, bestHit.oriNormal) > 0; // is ray entering or leaving refractive material?
			float nc = 1.0f;  // Index of Refraction air
			float nt = 1.5f;  // Index of Refraction glass/water
			float nnt = into ? nc / nt : nt / nc;  // IOR ratio of refractive materials
			float ddn = dot(ray.direction, bestHit.oriNormal);
			float cos2t = 1.0f - nnt * nnt * (1.f - ddn * ddn);

			if (cos2t < 0.0f) // total internal reflection 
			{
				bestHit.nextDir = reflect(ray.direction, bestHit.normal); //d = r.dir - 2.0f * n * dot(n, r.dir);
				hitPosition += bestHit.oriNormal * 0.01f;
			}
			else // cos2t > 0
			{
				// compute direction of transmission ray
				float3 tdir = normalize(ray.direction * nnt - bestHit.normal * ((into ? 1 : -1) * (ddn * nnt + sqrtf(cos2t))));

				float R0 = (nt - nc) * (nt - nc) / (nt + nc) * (nt + nc);
				float c = 1.f - (into ? -ddn : dot(tdir, bestHit.normal));
				float Re = R0 + (1.f - R0) * c * c * c * c * c;
				float Tr = 1 - Re; // Transmission
				float P = .25f + .5f * Re;
				float RP = Re / P;
				float TP = Tr / (1.f - P);

				// randomly choose reflection or transmission ray
				if (hiprand_uniform(randstate) < 0.25) // reflection ray
				{
					mask *= RP;
					bestHit.nextDir = reflect(ray.direction, bestHit.normal);
					hitPosition += bestHit.oriNormal * 0.01f;
				}
				else // transmission ray
				{
					mask *= TP;
					bestHit.nextDir = tdir; //r = Ray(x, tdir); 
					hitPosition += bestHit.oriNormal * 0.0005f; // epsilon must be small to avoid artefacts
				}
			}
		}

		// set up origin and direction of next path segment
		ray.origin = hitPosition;
		ray.direction = bestHit.nextDir;
	}

	// add radiance up to a certain ray depth
	// return accumulated color after all bounces are computed
	return accuIntensity;
}


__device__ unsigned char Color(float c)
{
	c = clamp(c, 0.0f, 1.0f);
	return int(c * 255.99) & 0xff;
}
__device__ float3 gammaCorrect(float3 c)
{
	float3 g;
	g.x = pow(c.x, 1 / 2.2f);
	g.y = pow(c.y, 1 / 2.2f);
	g.z = pow(c.z, 1 / 2.2f);
	return g;
}


__global__ void rand_init(int max_x, int max_y, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;
	int pixel_index = j * max_x + i;
	// Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1997 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(uchar4 *pos, float3* accumbuffer, hiprandState* randSt, int width, int height, int frameNum, int HashedFrameNum)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height)) 
		return;
	
	// unique id for the pixel
	int index = j * width + i;
	// create random number generator, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition, need refresh per frame
	hiprand_init(HashedFrameNum + index, 0, 0, &randState);
	float3 pixelColor = make_float3(0);
	// offset inside each pixel
	float offsetX = hiprand_uniform(&randState);	// get random float between (0, 1)
	float offsetY = hiprand_uniform(&randState);
	//float offsetX = m_rand(frameNum, make_float2(i, j));	// get random float between (0, 1)
	//float offsetY = m_rand(frameNum, make_float2(i, j));
	//if(index == 0 && frameNum < 100) printf("%f, %f\n", offsetX, offsetY);
	// uv(-0.5, 0.5)
	float2 uv = make_float2((i + offsetX) / width, (j + offsetY) / height) - make_float2(0.5f, 0.5f);
	Ray cam(camPos, normalize(make_float3(0.0f, 0.0f, -1.0f)));
	float3 screen = make_float3(uv.x * width, -uv.y * height, -500);
	float3 dir = normalize(screen - cam.origin);

	float intensity = radiance(Ray(cam.origin, dir), &randState, frameNum);
	pixelColor = make_float3(intensity);
	if (frameNum == 0) accumbuffer[index] = make_float3(0.0);	//init
	accumbuffer[index] += pixelColor;

	float3 tempCol = accumbuffer[index]/(float)frameNum;
	//tempCol = gammaCorrect(tempCol);

	// (0.0f, 1.0f) -> (0, 255)
	unsigned char r = Color(tempCol.x);
	unsigned char g = Color(tempCol.y);
	unsigned char b = Color(tempCol.z);
	//debug
	//unsigned char r = Color(dir.x);
	//unsigned char g = Color(dir.y);
	//unsigned char b = Color(dir.z);

	pos[index].w = 0;
	pos[index].x = r;
	pos[index].y = g;
	pos[index].z = b;
}

extern "C" void launch_kernel(uchar4* pos, float3* accumbuffer, hiprandState* randState, unsigned int w, unsigned int h, unsigned int frame) {

	//set thread number
	int tx = 16;
	int ty = 16;

	dim3 blocks(w / tx + 1, h / ty + 1);
	dim3 threads(tx, ty);
	render <<<blocks, threads >>> (pos, accumbuffer, randState, w, h, frame, WangHash(frame));

	hipDeviceSynchronize();
	checkCUDAError("kernel failed!");
}

